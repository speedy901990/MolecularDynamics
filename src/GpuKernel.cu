#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "GpuKernel.h"
#include "Log.h"
#include "CudaHelpers.h"
#include "GpuThread.h"

GpuKernel::GpuKernel() {

}

GpuKernel::~GpuKernel() {

}

int GpuKernel::allocateDeviceMemory(Structure * &atomsStructure, int deviceCount) {
  cout << "\t> Allocating device memory... ";
  
  if (false/*deviceCount == 1*/) {
    /*HANDLE_ERROR( hipMalloc( (void**)&devicePtr.inputAtomsStructure, sizeof(Structure) ) );
    HANDLE_ERROR( hipMalloc( (void**)&devicePtr.inputAtoms, sizeof(Atom) * atomsStructure->atomsCount ) );
    HANDLE_ERROR( hipMalloc( (void**)&devicePtr.outputAtomsStructure, sizeof(Structure) ) );
    HANDLE_ERROR( hipMalloc( (void**)&devicePtr.outputAtoms, sizeof(Atom) * atomsStructure->atomsCount ) );*/
  }
  else {
    devicePtr = new DevMemory[deviceCount];
    for (int i=0 ; i<deviceCount ; i++) {
      hipSetDevice(i);
      HANDLE_ERROR( hipMalloc( (void**)&devicePtr[i].inputAtomsStructure, sizeof(Structure) ) );
      HANDLE_ERROR( hipMalloc( (void**)&devicePtr[i].inputAtoms, sizeof(Atom) * atomsStructure[i].atomsCount ) );
      HANDLE_ERROR( hipMalloc( (void**)&devicePtr[i].outputAtomsStructure, sizeof(Structure) ) );
      HANDLE_ERROR( hipMalloc( (void**)&devicePtr[i].outputAtoms, sizeof(Atom) * atomsStructure[i].atomsCount ) );
    }
  }

  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::sendDataToDevice(Structure * &atomsStructure, int deviceCount) {
  cout << "\t> Sending data to device... " << flush;
  
  if (false/*deviceCount == 1*/) {
    /*HANDLE_ERROR( hipMemcpy( devicePtr.inputAtomsStructure, atomsStructure, sizeof(Structure), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( devicePtr.inputAtoms, atomsStructure->atoms, sizeof(Atom) * atomsStructure->atomsCount, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(devicePtr.inputAtomsStructure->atoms), &(devicePtr.inputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMemcpy( devicePtr.outputAtomsStructure, atomsStructure, sizeof(Structure), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( devicePtr.outputAtoms, atomsStructure->atoms, sizeof(Atom) * atomsStructure->atomsCount, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( &(devicePtr.outputAtomsStructure->atoms), &(devicePtr.outputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );*/
  }
  else {
    for (int i=0 ; i<deviceCount ; i++) {
      hipSetDevice(i);
      HANDLE_ERROR( hipMemcpy( devicePtr[i].inputAtomsStructure, atomsStructure + i, sizeof(Structure), hipMemcpyHostToDevice ) );
      HANDLE_ERROR( hipMemcpy( devicePtr[i].inputAtoms, atomsStructure[i].atoms, sizeof(Atom) * atomsStructure[i].atomsCount, hipMemcpyHostToDevice ) );
      HANDLE_ERROR( hipMemcpy( &(devicePtr[i].inputAtomsStructure->atoms), &(devicePtr[i].inputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );

      HANDLE_ERROR( hipMemcpy( devicePtr[i].outputAtomsStructure, atomsStructure + i, sizeof(Structure), hipMemcpyHostToDevice ) );
      HANDLE_ERROR( hipMemcpy( devicePtr[i].outputAtoms, atomsStructure[i].atoms, sizeof(Atom) * atomsStructure[i].atomsCount, hipMemcpyHostToDevice ) );
      HANDLE_ERROR( hipMemcpy( &(devicePtr[i].outputAtomsStructure->atoms), &(devicePtr[i].outputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );
    }
  }
  
  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::execute(Structure * structure, int devicesCount, bool displayOn) {
  cout << flush << "\t> Executing kernel... "<< flush;
  if (structure == NULL) {
    Log::instance()->toConsole(E_NULL_PTR, typeid(this).name(), __FUNCTION__, __LINE__, "Structure is NULL.");
    exit(EXIT_FAILURE);
  }
  
  this->structure = structure;

  if (displayOn) {
    executeDisplayOn();
  }
  else {
    if (devicesCount != 1)
      executeMultiGpu(devicesCount);
    else
      executeDisplayOff();
    cout << "\t...done!" << endl << flush;
  }

  return SUCCESS;
}

int GpuKernel::executeDisplayOn() {
  GpuDisplay::instance()->runAnimation(this);

  return SUCCESS;
}

int GpuKernel::executeMultiGpu(int deviceCount) {
  
  pthread_t * threads = new pthread_t[deviceCount];
  GpuThread * threadsData = new GpuThread[deviceCount];

  for (int i=0 ; i<deviceCount ; i++) {
    threadsData[i].kernel = this;
    threadsData[i].tid = i;
    threads[i] = startThread(executeGpuThreadKernel, (void *)threadsData);
  }
  
  for (int i=0 ; i<deviceCount ; i++)
    endThread( threads[i] );
  
  printf("------- TOTAL PERFORMANCE: --------");
  //TODO FIX segfault
  //for (int i=0 ; i<deviceCount ; i++)
  //  displayPerformanceResults(threadsData[i].performance);
  
  return SUCCESS;
}

PerformanceStatistics * GpuKernel::executeThreadKernel(int tid) {
  int mesh_width = structure[tid].dim.x;
  int mesh_height = structure[tid].dim.y;
  int threadsPerBlock = 1024;
  int blocksPerGrid = (mesh_width * mesh_width * mesh_width + threadsPerBlock - 1) / threadsPerBlock;
  dim3 block(threadsPerBlock, 1, 1);
  dim3 grid(blocksPerGrid, 1, 1);
  int nIter = 100;
  hipError_t error;
  float msecTotal = 0.0f;
  
  hipEvent_t start;
  handleTimerError(hipEventCreate(&start), START_CREATE);
  
  hipEvent_t stop;
  handleTimerError(hipEventCreate(&stop), STOP_CREATE);
  
  handleTimerError(hipEventRecord(start, NULL), START_RECORD);
  /*
  for (int i=0 ; i<nIter ; i++) {
    update_structure<<< grid, block >>>(devicePtr[tid].inputAtomsStructure, devicePtr[tid].outputAtomsStructure);
    MD_LJ_kernel<<< grid, block >>>(devicePtr[tid].inputAtomsStructure, devicePtr[i].outputAtomsStructure);
  }
  
  hipDeviceSynchronize();
  */
  handleTimerError(hipEventRecord(stop, NULL), STOP_RECORD);
  handleTimerError(hipEventSynchronize(stop), SYNCHRONIZE);
  handleTimerError(hipEventElapsedTime(&msecTotal, start, stop), ELAPSED_TIME);
  
  PerformanceStatistics * performance = new PerformanceStatistics(msecTotal, nIter, block, grid);
  displayPerformanceResults(performance);
  
  return performance;
}

int GpuKernel::executeDisplayOff() {
  int mesh_width = structure->dim.x;
  int mesh_height = structure->dim.y;
  int threadsPerBlock = 1024;
  int blocksPerGrid = (mesh_width * mesh_width * mesh_width + threadsPerBlock - 1) / threadsPerBlock;
  dim3 block(threadsPerBlock, 1, 1);
  dim3 grid(blocksPerGrid, 1, 1);
  int nIter = 1;
  hipError_t error;
  float msecTotal = 0.0f;

  hipEvent_t start;
  handleTimerError(hipEventCreate(&start), START_CREATE);

  hipEvent_t stop;
  handleTimerError(hipEventCreate(&stop), STOP_CREATE);
  
  handleTimerError(hipEventRecord(start, NULL), START_RECORD);

  for (int i=0 ; i<nIter ; i++) {
    update_structure<<< grid, block >>>(devicePtr[i].inputAtomsStructure, devicePtr[i].outputAtomsStructure);
    MD_LJ_kernel<<< grid, block >>>(devicePtr[i].inputAtomsStructure, devicePtr[i].outputAtomsStructure);
  }

  hipDeviceSynchronize();

  handleTimerError(hipEventRecord(stop, NULL), STOP_RECORD);
  handleTimerError(hipEventSynchronize(stop), SYNCHRONIZE);
  handleTimerError(hipEventElapsedTime(&msecTotal, start, stop), ELAPSED_TIME);

  PerformanceStatistics * performance = new PerformanceStatistics(msecTotal, nIter, block, grid);
  displayPerformanceResults(performance);
  
  return SUCCESS;
}

void GpuKernel::displayPerformanceResults(PerformanceStatistics *p) {
  float msecPerSimulation = p->msecTotal / p->nIter;
  double flopsPerSimulation = 55.0 * structure->atomsCount * structure->atomsCount + 10.0 * structure->atomsCount + 2 * (structure->atomsCount + 256 - 1 )/ 256;
  double gigaFlops = (flopsPerSimulation * 1.0e-9f) / (msecPerSimulation / 1000.0f);
  printf("\n\t\tPerformance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
	 gigaFlops,
	 msecPerSimulation,
	 flopsPerSimulation,
	 p->block.x * p->block.y);
}

void GpuKernel::executeInsideGlutLoop(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (mesh_width * mesh_width * mesh_width + threadsPerBlock - 1) / threadsPerBlock;
  dim3 block(threadsPerBlock, 1, 1);
  dim3 grid(blocksPerGrid, 1, 1);

  //  update_structure_and_display<<< grid, block >>>(pos, devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure);
  update_structure_and_display<<< grid, block >>>(pos, devicePtr[0].inputAtomsStructure, devicePtr[0].outputAtomsStructure);
  //  MD_LJ_kernel<<< grid, block >>>(devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure, time);
  MD_LJ_kernel<<< grid, block >>>(devicePtr[0].inputAtomsStructure, devicePtr[0].outputAtomsStructure, time);
  hipDeviceSynchronize();
}

int GpuKernel::getDataFromDevice(Structure *&atomsStructure, int deviceCount) {
  Structure * tmpOutputData = new Structure();
  Atom * atoms = new Atom[atomsStructure->atomsCount];

  cout << "\t> Receiving data from device... ";

  for (int i=0 ; i<1 ; i++) {
    HANDLE_ERROR( hipMemcpy( tmpOutputData, devicePtr[i].outputAtomsStructure, sizeof(Structure), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( /*tmpOutputData->atoms*/atoms, /*devicePtr.outputAtomsStructure->atoms*/devicePtr[i].outputAtoms, sizeof(Atom) * atomsStructure[i].atomsCount, hipMemcpyDeviceToHost ) );
  }
  cout << "done!" << endl;
  
  /*cout << "Data:" << endl;
  for (int i=0 ; i<atomsStructure->atomsCount ; i++) {
    cout << "Atom " << i << " x=" << atoms[i].pos.x << " y=" << atoms[i].pos.y << " z=" << atoms[i].pos.z
	 <<endl;//	 << " gradientX=" << atoms[i].gradientX << " gradientY=" << atoms[i].gradientY << " gradientZ=" << atoms[i].gradientZ << " force=" << atoms[i].force <<  endl;
  }
  */
  return SUCCESS;
}

int GpuKernel::clearDeviceMemory(int devicesCount) {
  if (false/*devicesCount == 1*/) {
    /*HANDLE_ERROR( hipFree( devicePtr.inputAtomsStructure ) );
    HANDLE_ERROR( hipFree( devicePtr.outputAtomsStructure ) );
    HANDLE_ERROR( hipFree( devicePtr.inputAtoms ) );
    HANDLE_ERROR( hipFree( devicePtr.outputAtoms ) );*/
  }
  else {
    for (int i=0 ; i<devicesCount ; i++) {
      HANDLE_ERROR( hipFree( devicePtr[i].inputAtomsStructure ) );
      HANDLE_ERROR( hipFree( devicePtr[i].outputAtomsStructure ) );
      HANDLE_ERROR( hipFree( devicePtr[i].inputAtoms ) );
      HANDLE_ERROR( hipFree( devicePtr[i].outputAtoms ) );
    }
  }

  hipDeviceReset();

  return SUCCESS;
}
