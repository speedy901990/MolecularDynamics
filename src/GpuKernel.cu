#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "GpuKernel.h"
#include "Log.h"
#include "CudaHelpers.h"

GpuKernel::GpuKernel() {

}

GpuKernel::~GpuKernel() {

}

int GpuKernel::allocateDeviceMemory(Structure * &atomsStructure) {
  cout << "\t> Allocating device memory... ";
  
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.inputAtomsStructure, sizeof(Structure) ) );
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.inputAtoms, sizeof(Atom) * atomsStructure->atomsCount ) );
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.outputAtomsStructure, sizeof(Structure) ) );
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.outputAtoms, sizeof(Atom) * atomsStructure->atomsCount ) );
  
  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::sendDataToDevice(Structure * &atomsStructure) {
  cout << "\t> Sending data to device... " << flush;
  
  HANDLE_ERROR( hipMemcpy( devicePtr.inputAtomsStructure, atomsStructure, sizeof(Structure), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( devicePtr.inputAtoms, atomsStructure->atoms, sizeof(Atom) * atomsStructure->atomsCount, hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( &(devicePtr.inputAtomsStructure->atoms), &(devicePtr.inputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );

  HANDLE_ERROR( hipMemcpy( devicePtr.outputAtomsStructure, atomsStructure, sizeof(Structure), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( devicePtr.outputAtoms, atomsStructure->atoms, sizeof(Atom) * atomsStructure->atomsCount, hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( &(devicePtr.outputAtomsStructure->atoms), &(devicePtr.outputAtoms), sizeof(Atom *), hipMemcpyHostToDevice ) );
  
  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::execute(bool displayOn) {
  cout << flush << "\t> Executing kernel... "<< flush;
  if (displayOn) {
    executeDisplayOn();
  }
  else {
    executeDisplayOff();
    cout << "done!" << endl << flush;
  }

  return SUCCESS;
}

int GpuKernel::executeDisplayOn() {
  GpuDisplay::instance()->runAnimation(this);

  return SUCCESS;
}

int GpuKernel::executeDisplayOff() {
  //  atomsStructureTest<<<1,1>>>( devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure);
  MD_LJ_kernel_no_visual<<<1,1>>>(devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure);

  return SUCCESS;
}

void GpuKernel::executeInsideGlutLoop(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time) {
  dim3 block(2, 2, 1);
  dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
  //vbo_MD_kernel<<<grid, block>>>(pos, devicePtr.inputAtomsStructure, time);
  //vbo_MD_kernel<<<1,1>>>(pos, devicePtr.inputAtomsStructure, time);
  MD_LJ_kernel<<<grid,block>>>(pos, devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure, time);
  //MD_LJ_kernel<<<1,1>>>(pos, devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure, time);
}

int GpuKernel::getDataFromDevice(Structure *&atomsStructure) {
  Structure * tmpOutputData = new Structure();
  Atom * atoms = new Atom[atomsStructure->atomsCount];

  cout << "\t> Receiving data from device... ";
  // TODO@@@@@@@@@@
  HANDLE_ERROR( hipMemcpy( tmpOutputData, devicePtr.outputAtomsStructure, sizeof(Structure), hipMemcpyDeviceToHost ) );
  HANDLE_ERROR( hipMemcpy( /*tmpOutputData->atoms*/atoms, /*devicePtr.outputAtomsStructure->atoms*/devicePtr.outputAtoms, sizeof(Atom) * atomsStructure->atomsCount, hipMemcpyDeviceToHost ) );

  cout << "done!" << endl;

  cout << "Data:" << endl;
  for (int i=0 ; i<atomsStructure->atomsCount ; i++) {
    cout << "Atom " << i << " x=" << atoms[i].pos.x << " y=" << atoms[i].pos.y << " z=" << atoms[i].pos.z
	 <<endl;//	 << " gradientX=" << atoms[i].gradientX << " gradientY=" << atoms[i].gradientY << " gradientZ=" << atoms[i].gradientZ << " force=" << atoms[i].force <<  endl;
  }

  return SUCCESS;
}

int GpuKernel::clearDeviceMemory() {
  HANDLE_ERROR( hipFree( devicePtr.inputAtomsStructure ) );
  HANDLE_ERROR( hipFree( devicePtr.outputAtomsStructure ) );
  HANDLE_ERROR( hipFree( devicePtr.inputAtoms ) );
  HANDLE_ERROR( hipFree( devicePtr.outputAtoms ) );

  hipDeviceReset();

  return SUCCESS;
}

