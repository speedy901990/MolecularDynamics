#include <hip/hip_runtime.h>

#include "GpuKernel.h"
#include "Log.h"
#include "CudaHelpers.h"

GpuKernel::GpuKernel() {

}

GpuKernel::~GpuKernel() {

}

int GpuKernel::allocateDeviceMemory() {
  cout << "\t> Allocating device memory... ";
  
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.inputAtomsStructure, sizeof(Structure) ) );
  HANDLE_ERROR( hipMalloc( (void**)&devicePtr.outputAtomsStructure, sizeof(Structure) ) );
  
  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::sendDataToDevice(Structure * atomsStructure) {
  cout << "\t> Sending data to device... ";
  
  HANDLE_ERROR( hipMemcpy( devicePtr.inputAtomsStructure, atomsStructure, sizeof(Structure), hipMemcpyHostToDevice ) );
  
  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::executeKernel() {
  cout << "\t> Executing kernel... ";

  atomsStructureTest<<<1,1>>>( devicePtr.inputAtomsStructure, devicePtr.outputAtomsStructure);

  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::getDataFromDevice() {
  Structure * tmpOutputData = new Structure();
  cout << "\t> Receiving data from device... ";

  HANDLE_ERROR( hipMemcpy( tmpOutputData, devicePtr.outputAtomsStructure, sizeof(Structure), hipMemcpyDeviceToHost ) );

  cout << "done!" << endl;

  return SUCCESS;
}

int GpuKernel::clearDeviceMemory() {
  HANDLE_ERROR( hipFree( devicePtr.inputAtomsStructure ) );
  HANDLE_ERROR( hipFree( devicePtr.outputAtomsStructure ) );

  hipDeviceReset();
  return SUCCESS;
}

