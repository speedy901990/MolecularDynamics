#include "hip/hip_runtime.h"
#include "CudaHelpers.h"
#include "GpuThread.h"

//------------------- Kernels ----------------------------------
__global__ void atomsStructureTest( Structure * input, Structure * output) {
  //    int tid = threadIdx.x + blockIdx.x * blockDim.x;
  output->atomsCount = input->atomsCount;
}


__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
}

__global__ void update_structure(Structure *input, Structure *output) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  input->atomsCount = output->atomsCount;

  #pragma unroll
  for (int i=tid ; i<input->atomsCount ; i+=blockDim.x * gridDim.x) {
    //for (int i=0 ; i<input->atomsCount ; i++) {
    input->atoms[i].pos.x = output->atoms[i].pos.x;
    input->atoms[i].pos.y = output->atoms[i].pos.y;
    input->atoms[i].pos.z = output->atoms[i].pos.z;
    /*input->atoms[i].force = output->atoms[i].force;
    input->atoms[i].acceleration = output->atoms[i].acceleration;
    input->atoms[i].status = output->atoms[i].status;
    input->atoms[i].fixed = output->atoms[i].fixed;*/
  }
}

__global__ void update_structure_and_display(float4 *pos, Structure *input, Structure *output) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float u, v, w;

  input->atomsCount = output->atomsCount;

  #pragma unroll
  for (int i=tid ; i<input->atomsCount ; i+=blockDim.x * gridDim.x) {
    //for (int i=0 ; i<input->atomsCount ; i++) {
    input->atoms[i].pos.x = output->atoms[i].pos.x;
    input->atoms[i].pos.y = output->atoms[i].pos.y;
    input->atoms[i].pos.z = output->atoms[i].pos.z;
    /*input->atoms[i].force = output->atoms[i].force;
    input->atoms[i].acceleration = output->atoms[i].acceleration;
    input->atoms[i].status = output->atoms[i].status;
    input->atoms[i].fixed = output->atoms[i].fixed;*/

    u = input->atoms[i].pos.x * 0.1f;
    w = input->atoms[i].pos.y * 0.1f;
    v = input->atoms[i].pos.z * 0.1f;
    pos[i] = make_float4(u, w, v, 1.0f);
  }
}

__global__ void MD_LJ_kernel(Structure *input, Structure *output, float time) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int atomIndexStart = tid;
  int atomIndexEnd = input->atomsCount;
  float force[3] = {0.0f, 0.0f, 0.0f};
  
  // COMPUTING
  register float dX = 0.0f;
  register float dY = 0.0f;
  register float dZ = 0.0f;
  register float x = 0.0f, y = 0.0f, z = 0.0f;
  register float distance = 0.0f;
  register float potential = 0.0f;
  register float deltaTimeSquare = 0.0025;//pow(0.05f, 2);

  #pragma unroll
  for (register int i=atomIndexStart ; i<atomIndexEnd ; i += blockDim.x * gridDim.x) {
    force[0] = 0.0f;
    force[1] = 0.0f;
    force[2] = 0.0f;
    
    #pragma unroll
    for (register int j=0 ; j<input->atomsCount ; j++) {
      if (i == j)
	continue;
      
      dX = input->atoms[j].pos.x - input->atoms[i].pos.x;
      dY = input->atoms[j].pos.y - input->atoms[i].pos.y;
      dZ = input->atoms[j].pos.z - input->atoms[i].pos.z;
      distance = sqrtf(pow(dX, 2) + pow(dY, 2) + pow(dZ, 2));
      
      if (distance >= 2.5)
	continue;
      
      potential = 4.0f * (pow((1.0f/distance), 12) -  pow((1.0f/distance), 6) );
      if (potential > 50.0f)
	continue;

      force[0] += -(dX / distance) * potential;// * input->atoms[i].force;
      force[1] += -(dY / distance) * potential;// * input->atoms[i].force;
      force[2] += -(dZ / distance) * potential;// * input->atoms[i].force;
  }
    output->atoms[i].pos.x = input->atoms[i].pos.x + 0.5 * force[0] * deltaTimeSquare;
    output->atoms[i].pos.y = input->atoms[i].pos.y + 0.5 * force[1] * deltaTimeSquare;
    output->atoms[i].pos.z = input->atoms[i].pos.z + 0.5 * force[2] * deltaTimeSquare;
 }
}

// ERROR handling-----------------------------------------------------------

void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

// Threading for multi GPU support------------------------------------------
//typedef void *(*CUT_THREADROUTINE)(void *);

pthread_t startThread(CUT_THREADROUTINE func, void * data){
    pthread_t thread;
    pthread_create(&thread, NULL, func, data);
    return thread;
}

void endThread(pthread_t thread) {
    pthread_join(thread, NULL);
}

void * executeGpuThreadKernel(void * threadData) {
  GpuThread * data = (GpuThread *) threadData;
  data->performance = data->kernel->executeThreadKernel(data->tid);
}
// Other helper methodes-----------------------------------------------------
void displayAvailableDevices() {
    int deviceCount;
    hipError_t error;
    hipDeviceProp_t deviceProp;

    error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
        printf("hipGetDeviceCount returned error code %d, line(%d)\n", error, __LINE__);

    printf("Available devices: %d\n", deviceCount);

    for (int i=0 ; i<deviceCount ; i++) {
        error = hipGetDeviceProperties(&deviceProp, i);
        if (error != hipSuccess)
            printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
}

void displayChosenDevices(int * devicesID, int devicesCount) {
  hipError_t error;
  hipDeviceProp_t deviceProp;
  printf("Chosen devices: %d\n", devicesCount);

  for (int i=0 ; i<devicesCount ; i++) {
        error = hipGetDeviceProperties(&deviceProp, devicesID[i]);
        if (error != hipSuccess)
            printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devicesID[i], deviceProp.name, deviceProp.major, deviceProp.minor);
    }
}

void getDevices(int * &devicesID, int &devicesCount) {
  hipError_t error;
  int devicesLimit;
  error = hipGetDeviceCount(&devicesLimit);
  if (error != hipSuccess) {
    printf("hipGetDeviceCount returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (devicesCount > devicesLimit) {
    printf("ERR: devicesCount cannot be larger than devicesLimit returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

      // TODO
  for (int i=0 ; i<devicesCount ; i++) {
    error = hipSetDevice(devicesID[i]);;
    if (error != hipSuccess) {
      printf("hipSetDevice returned error code, %d, line(%d) - no such device\n", error, __LINE__);
      exit(EXIT_SUCCESS);
    }
  } 
}

void handleTimerError(hipError_t error, int type) {
  if (error == hipSuccess)
    return;

  switch (type) {
  case START_CREATE:
    fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
    break;
  case STOP_CREATE:
    fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
    break;
  case START_RECORD:
    fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
    break;
  case STOP_RECORD:
    fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
    break;
  case SYNCHRONIZE:
    fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
    break;
  case ELAPSED_TIME:
    fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
    break;
  default:
    fprintf(stderr, "Unknown error!\n");
    break;
  }
  
  exit(EXIT_FAILURE);
}
